#include "buffer.cuh"

#include "kernel/kernel.cuh"

namespace ghp
{

//----------------//
//    Graphite    //
//----------------//

void initGraphite() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
}

void destroyGraphite() {
    BufferRegister::destroyInstance();
}

void draw() {

    BufferRegister* bufferRegister = BufferRegister::getInstance();
    if(bufferRegister->getBindedFrameBufferID() > 0) {

        // FrameBuffer
        Ptr<FrameBuffer> bindedFrameBuffer = bufferRegister->getBindedFrameBuffer();
        uint8_t* frameBuffer = (uint8_t*)bindedFrameBuffer->getBuffer();
        
        const unsigned int width = bindedFrameBuffer->getWidth();
        const unsigned int height = bindedFrameBuffer->getHeight();

        // Vertex Buffer
        float* vertexBuffer = nullptr;
        if(bufferRegister->getBindedVertexBufferID() > 0)
            vertexBuffer = (float*)bufferRegister->getBindedVertexBuffer()->getBuffer();

        // Index Buffer
        unsigned int* indexBuffer = nullptr;
        if(bufferRegister->getBindedIndexBufferID() > 0)
            indexBuffer = (unsigned int*)bufferRegister->getBindedIndexBuffer()->getBuffer();

        // Draw kernel
        dim3 threadsPerBlock(16, 16);
        dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                        (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

        kernel<<<blocksPerGrid, threadsPerBlock>>>(frameBuffer, vertexBuffer, indexBuffer, width, height);
        hipDeviceSynchronize();
    }
}

//------------------//
//  BufferRegister  //
//------------------//

BufferRegister* BufferRegister::getInstance() {
    if(instance == nullptr)
        instance = new BufferRegister();

    return instance;
}

void BufferRegister::destroyInstance() {
    if (instance != nullptr) {
        delete instance;
        instance = nullptr;
    }
}

BufferRegister* BufferRegister::instance = nullptr;

//-------------//
//   Buffer    //
//-------------//

Buffer::Buffer(unsigned int _id, size_t _size)
    : id(_id), size(_size) {
    hipMalloc((void**)&buffer, size);
    check_cuda_error("Buffer::Buffer hipMalloc");
}

Buffer::~Buffer() {
    hipFree(buffer);
    check_cuda_error("Buffer::~Buffer hipFree");
}

Buffer::Buffer(const Buffer& buff)
    : id(buff.id), size(buff.size) {
    if (buff.buffer) {
        hipMalloc((void**)&buffer, size);
        check_cuda_error("Buffer::Buffer (copy) hipMalloc");
        hipMemcpy(buffer, buff.buffer, size, hipMemcpyDeviceToDevice);
        check_cuda_error("Buffer::Buffer (copy) hipMemcpy");
    }
}

Buffer::Buffer(Buffer&& buff) noexcept
    : buffer(buff.buffer), id(buff.id), size(buff.size) {
    buff.buffer = nullptr;
    buff.size = 0;
    buff.id = 0;
}

Buffer& Buffer::operator=(const Buffer& buff) {

    if (this != &buff) {
 
        hipFree(buffer);
        check_cuda_error("Buffer::operator= hipFree");


        id = buff.id;
        size = buff.size;

        if (buff.buffer) {
            hipMalloc((void**)&buffer, size);
            check_cuda_error("Buffer::operator= hipMalloc");
            hipMemcpy(buffer, buff.buffer, size, hipMemcpyDeviceToDevice);
            check_cuda_error("Buffer::operator= hipMemcpy");
        } else {
            buffer = nullptr;
        }
    }

    return *this;
}

Buffer& Buffer::operator=(Buffer&& buff) noexcept {

    if (this != &buff) {

        hipFree(buffer);
        check_cuda_error("Buffer::operator= (move) hipFree");

        buffer = buff.buffer;
        id = buff.id;
        size = buff.size;

        buff.buffer = nullptr;
        buff.size = 0;
        buff.id = 0;
    }

    return *this;
}

//-----------------//
//   FrameBuffer   //
//-----------------//

FrameBuffer::FrameBuffer(unsigned int id, unsigned int _width, unsigned int _height)
    : Buffer(id, 1 * width * height * 3), width(_width), height(_height) {
}

FrameBuffer::FrameBuffer(unsigned int _width, unsigned int _height)
    : FrameBuffer(0, width, height) {
}

FrameBuffer::FrameBuffer(const FrameBuffer& frameBuffer)
    : Buffer(frameBuffer), width(frameBuffer.width),
    height(frameBuffer.height) {
}

FrameBuffer::FrameBuffer(FrameBuffer&& frameBuffer) noexcept
    : Buffer(std::move(frameBuffer)), width(frameBuffer.width),
    height(frameBuffer.height) {
    frameBuffer.width = 0;
    frameBuffer.height = 0;
}

FrameBuffer& FrameBuffer::operator=(const FrameBuffer& frameBuffer) {

    if (this != &frameBuffer) {

        Buffer::operator=(frameBuffer);

        width = frameBuffer.width;
        height = frameBuffer.height;
    }

    return *this;
}

FrameBuffer& FrameBuffer::operator=(FrameBuffer&& frameBuffer) noexcept {

    if (this != &frameBuffer) {

        Buffer::operator=(std::move(frameBuffer));

        width = frameBuffer.width;
        height = frameBuffer.height;

        frameBuffer.width = 0;
        frameBuffer.height = 0;
    }

    return *this;
}

Ptr<FrameBuffer> FrameBuffer::New(unsigned int width, unsigned int height) {

    BufferRegister* bufferRegister = BufferRegister::getInstance();

    int id = bufferRegister->getFrameBuffers().size() + 1;
    Ptr<FrameBuffer> frameBuffer = std::make_shared<FrameBuffer>(id, width, height);
    bufferRegister->addFrameBuffer(frameBuffer);

    return frameBuffer;
}

void FrameBuffer::bind() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
    bufferRegister->bindFbo(id);
}

void FrameBuffer::unbind() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
    bufferRegister->bindFbo(0);
}

//------------------//
//   VertexBuffer   //
//------------------//

VertexBuffer::VertexBuffer(unsigned int id, float* data, size_t size, const Attributes& _attr) 
    : Buffer(id, size), attributes(_attr) {
    hipMemcpy(buffer, data, size, hipMemcpyHostToDevice);
    check_cuda_error("VertexBuffer::VertexBuffer hipMemcpy");
}

VertexBuffer::VertexBuffer(const VertexBuffer& vertexBuffer) 
    : Buffer(vertexBuffer), attributes(vertexBuffer.attributes) {
}

VertexBuffer::VertexBuffer(VertexBuffer&& vertexBuffer) noexcept 
    : Buffer(std::move(vertexBuffer)), attributes(std::move(vertexBuffer.attributes)) {
    vertexBuffer.attributes = {};
}

VertexBuffer& VertexBuffer::operator=(const VertexBuffer& vertexBuffer) {

    if(this != &vertexBuffer) {
        Buffer::operator=(vertexBuffer);
        attributes = vertexBuffer.attributes;
    }

    return *this;
}

VertexBuffer& VertexBuffer::operator=(VertexBuffer&& vertexBuffer) noexcept {

    if(this != &vertexBuffer) {
        Buffer::operator=(std::move(vertexBuffer));
        attributes = std::move(vertexBuffer.attributes);
    }

    return *this;
}

Ptr<VertexBuffer> VertexBuffer::New(float* data, size_t size, const Attributes& attributes) {

    BufferRegister* bufferRegister = BufferRegister::getInstance();

    int id = bufferRegister->getVertexBuffers().size() + 1;
    Ptr<VertexBuffer> vertexBuffer = std::make_shared<VertexBuffer>(id, data, size, attributes);
    bufferRegister->addVertexBuffer(vertexBuffer);

    return vertexBuffer;
}

void VertexBuffer::bind() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
    bufferRegister->bindVbo(id);
}

void VertexBuffer::unbind() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
    bufferRegister->bindVbo(0);
}

//-----------------//
//   IndexBuffer   //
//-----------------//

IndexBuffer::IndexBuffer(unsigned int id, unsigned int* indices, size_t size) 
    : Buffer(id, size) {
    hipMemcpy(buffer, indices, size, hipMemcpyHostToDevice);
    check_cuda_error("IndexBuffer::IndexBuffer hipMemcpy");
}

IndexBuffer::IndexBuffer(const IndexBuffer& indexBuffer) 
    : Buffer(indexBuffer) {
}

IndexBuffer::IndexBuffer(IndexBuffer&& indexBuffer) noexcept 
    : Buffer(std::move(indexBuffer)) {
}

IndexBuffer& IndexBuffer::operator=(const IndexBuffer& indexBuffer) {

    if(this != &indexBuffer) {
        Buffer::operator=(indexBuffer);
    }

    return *this;
}

IndexBuffer& IndexBuffer::operator=(IndexBuffer&& indexBuffer) noexcept {

    if(this != &indexBuffer) {
        Buffer::operator=(std::move(indexBuffer));
    }

    return *this;
}

Ptr<IndexBuffer> IndexBuffer::New(unsigned int* indices, size_t size) {

    BufferRegister* bufferRegister = BufferRegister::getInstance();

    int id = bufferRegister->getIndexBuffers().size() + 1;
    Ptr<IndexBuffer> indexBuffer = std::make_shared<IndexBuffer>(id, indices, size);
    bufferRegister->addIndexBuffer(indexBuffer);

    return indexBuffer;
}

void IndexBuffer::bind() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
    bufferRegister->bindIbo(id);
}

void IndexBuffer::unbind() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
    bufferRegister->bindIbo(0);
}

//------------//
//    CUDA    //
//------------//

void check_cuda_error(const char* message) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error after " << message << ": " << hipGetErrorString(error) << std::endl;
        exit(-1);
    }
}

}