#include "buffer.cuh"

#include <kernel/kernel.cuh>

namespace ghp
{

//----------------//
//    Graphite    //
//----------------//

void initGraphite() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
}

void destroyGraphite() {
    BufferRegister::destroyInstance();
}

void draw() {

    BufferRegister* bufferRegister = BufferRegister::getInstance();

    if(bufferRegister->getBindedFrameBufferID() > 0) {
        Ptr<FrameBuffer> bindedFrameBuffer = bufferRegister->getBindedFrameBuffer();
        bindedFrameBuffer->draw();
    }
}

//------------------//
//  BufferRegister  //
//------------------//

BufferRegister* BufferRegister::getInstance() {
    if(instance == nullptr)
        instance = new BufferRegister();

    return instance;
}

void BufferRegister::destroyInstance() {
    if (instance != nullptr) {
        delete instance;
        instance = nullptr;
    }
}

BufferRegister* BufferRegister::instance = nullptr;

//-------------//
//   Buffer    //
//-------------//

Buffer::Buffer(unsigned int _id, size_t _size)
    : id(_id), size(_size) {
    hipMalloc((void**)&buffer, size);
    check_cuda_error("Buffer::Buffer hipMalloc");
}

Buffer::~Buffer() {
    hipFree(buffer);
    check_cuda_error("Buffer::~Buffer hipFree");
}

Buffer::Buffer(const Buffer& buff)
    : id(buff.id), size(buff.size) {
    if (buff.buffer) {
        hipMalloc((void**)&buffer, size);
        check_cuda_error("Buffer::Buffer (copy) hipMalloc");
        hipMemcpy(buffer, buff.buffer, size, hipMemcpyDeviceToDevice);
        check_cuda_error("Buffer::Buffer (copy) hipMemcpy");
    }
}

Buffer::Buffer(Buffer&& buff) noexcept
    : buffer(buff.buffer), id(buff.id), size(buff.size) {
    buff.buffer = nullptr;
    buff.size = 0;
    buff.id = 0;
}

Buffer& Buffer::operator=(const Buffer& buff) {

    if (this != &buff) {
 
        hipFree(buffer);
        check_cuda_error("Buffer::operator= hipFree");


        id = buff.id;
        size = buff.size;

        if (buff.buffer) {
            hipMalloc((void**)&buffer, size);
            check_cuda_error("Buffer::operator= hipMalloc");
            hipMemcpy(buffer, buff.buffer, size, hipMemcpyDeviceToDevice);
            check_cuda_error("Buffer::operator= hipMemcpy");
        } else {
            buffer = nullptr;
        }
    }

    return *this;
}

Buffer& Buffer::operator=(Buffer&& buff) noexcept {

    if (this != &buff) {

        hipFree(buffer);
        check_cuda_error("Buffer::operator= (move) hipFree");

        buffer = buff.buffer;
        id = buff.id;
        size = buff.size;

        buff.buffer = nullptr;
        buff.size = 0;
        buff.id = 0;
    }

    return *this;
}

//-----------------//
//   FrameBuffer   //
//-----------------//

FrameBuffer::FrameBuffer(unsigned int id, unsigned int _width, unsigned int _height)
    : Buffer(id, size()), width(_width), height(_height) {
}

FrameBuffer::FrameBuffer(unsigned int _width, unsigned int _height)
    : FrameBuffer(0, width, height) {
}

FrameBuffer::FrameBuffer(const FrameBuffer& frameBuffer)
    : Buffer(frameBuffer), width(frameBuffer.width),
    height(frameBuffer.height) {
}

FrameBuffer::FrameBuffer(FrameBuffer&& frameBuffer) noexcept
    : Buffer(std::move(frameBuffer)), width(frameBuffer.width),
    height(frameBuffer.height) {
    frameBuffer.width = 0;
    frameBuffer.height = 0;
}

FrameBuffer& FrameBuffer::operator=(const FrameBuffer& frameBuffer) {

    if (this != &frameBuffer) {

        Buffer::operator=(frameBuffer);

        width = frameBuffer.width;
        height = frameBuffer.height;
    }

    return *this;
}

FrameBuffer& FrameBuffer::operator=(FrameBuffer&& frameBuffer) noexcept {

    if (this != &frameBuffer) {

        Buffer::operator=(std::move(frameBuffer));

        width = frameBuffer.width;
        height = frameBuffer.height;

        frameBuffer.width = 0;
        frameBuffer.height = 0;
    }
    
    return *this;
}

Ptr<FrameBuffer> FrameBuffer::New(unsigned int width, unsigned int height) {

    BufferRegister* bufferRegister = BufferRegister::getInstance();

    int id = bufferRegister->getFrameBuffers().size() + 1;
    Ptr<FrameBuffer> frameBuffer = std::make_shared<FrameBuffer>(id, width, height);
    bufferRegister->addFrameBuffer(frameBuffer);

    return frameBuffer;
}

void FrameBuffer::bind() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
    bufferRegister->bindFbo(id);
}

void FrameBuffer::unbind() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
    bufferRegister->bindFbo(0);
}

void FrameBuffer::draw() {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    kernel<<<blocksPerGrid, threadsPerBlock>>>((uint8_t*)buffer, width, height);
    hipDeviceSynchronize();
}

//------------//
//    CUDA    //
//------------//

void check_cuda_error(const char* message) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error after " << message << ": " << hipGetErrorString(error) << std::endl;
        exit(-1);
    }
}

}