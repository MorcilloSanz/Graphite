#include <iostream>
#include <cstdint>

//#define STB_IMAGE_IMPLEMENTATION
#include <vendor/stb_image.h>

//#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <vendor/stb_image_write.h>

#include "graphics/buffer.cuh"
#include "graphics/renderer.cuh"

#include "math/linalg.cuh"
#include "math/transform.cuh"

#include "scene/scene.cuh"
#include "scene/model.cuh"

using namespace gph;

int main() {

    hipDeviceReset();

    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    std::cout << "Free Memory (GPU): " << freeMem / (1024 * 1024) << " MB" << std::endl;
    std::cout << "Total Memory (GPU): " << totalMem / (1024 * 1024) << " MB" << std::endl;

    // Renderer
    constexpr unsigned int width = 1600;
    constexpr unsigned int height = 900;
    
    Renderer renderer(width, height);

    // Sky
    int skyWidth, skyHeight, skyChannels;

    stbi_set_flip_vertically_on_load(1);
    float* skyData = stbi_loadf("C:/Users/amorc/Documents/Dev/3D/360images/aerodynamics_workshop_2k.hdr", &skyWidth, &skyHeight, &skyChannels, STBI_rgb_alpha);

    TextureHDR::Ptr sky = TextureHDR::New(skyData, skyWidth, skyHeight);
    renderer.setSky(sky);

    stbi_image_free(skyData);

    // Model
    Model::Ptr model = Model::fromFile("c:/Users/amorc/Documents/Dev/3D/models/glTF-Sample-Models/2.0/DamagedHelmet/glTF/DamagedHelmet.gltf");

    // Draw call
    renderer.clear();

    mat4<float> modelMatrix = rotationX<float>(M_PI / 2.0f) * rotationZ<float>(M_PI / 4.0f);
    mat4<float> viewMatrix = scale<float>(vec3<float>(0.75f));

    Uniforms uniforms(modelMatrix, viewMatrix);

    renderer.setUniforms(uniforms);
    renderer.draw(model);

    // CPU image
    uint8_t* bufferCPU = new uint8_t[renderer.getFrameBuffer()->size];

    hipMemcpy(bufferCPU, renderer.getFrameBuffer()->buff, renderer.getFrameBuffer()->size, hipMemcpyDeviceToHost);
    stbi_write_png("output.png", width, height, STBI_rgb, bufferCPU, width * STBI_rgb);
    
    delete[] bufferCPU;

    return 0;
}