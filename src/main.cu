#include <iostream>
#include <cstdint>

#define STB_IMAGE_IMPLEMENTATION
#include <vendor/stb_image.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <vendor/stb_image_write.h>

#include "graphics/buffer.cuh"
#include "graphics/renderer.cuh"

#include "math/linalg.cuh"
#include "math/transform.cuh"

using namespace gph;

int main() {

    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    std::cout << "Free Memory (GPU): " << freeMem / (1024 * 1024) << " MB" << std::endl;
    std::cout << "Total Memory (GPU): " << totalMem / (1024 * 1024) << " MB" << std::endl;

    // Renderer
    constexpr unsigned int width = 1080;
    constexpr unsigned int height = 720;
    
    Renderer renderer(width, height);

    size_t framebufferMem = sizeof(uint8_t) * width * height * 3;
    std::cout << "Frame Buffer Mem (GPU): " << static_cast<float>(framebufferMem) / (1024 * 1024) << "MB" << std::endl;

    // Vertex Buffer: x y z r g b nx ny nz uvx uvy
    float vertices[] = {
        -0.5f, -0.5f, -0.5f,  1.0f, 0.0f, 0.0f,  0.0f, 0.0f, -1.0f,  0.0f, 0.0f,
         0.5f, -0.5f, -0.5f,  0.0f, 1.0f, 0.0f,  0.0f, 0.0f, -1.0f,  1.0f, 0.0f,
         0.5f, -0.5f,  0.5f,  0.0f, 0.0f, 1.0f,  0.0f, 0.0f, -1.0f,  1.0f, 1.0f,
        -0.5f, -0.5f,  0.5f,  0.0f, 1.0f, 1.0f,  0.0f, 0.0f, -1.0f,  0.0f, 1.0f,
         0.0f,  0.5f,  0.0f,  1.0f, 0.5f, 0.5f,  0.0f, 1.0f,  0.0f,  0.5f, 0.5f
    };

    Buffer<float> vertexBuffer(vertices, sizeof(vertices));

    // Index buffer
    unsigned int indices[] = {
        0, 1, 4, 1, 2, 4,
        2, 3, 4, 3, 0, 4,
        0, 1, 2, 0, 2, 3
    };
    
    Buffer<unsigned int> indexBuffer(indices, sizeof(indices));

    // Sky
    int skyWidth, skyHeight, skyChannels;

    stbi_set_flip_vertically_on_load(1);
    unsigned char* skyData = stbi_load("C:/Users/amorc/Desktop/sky.png", &skyWidth, &skyHeight, &skyChannels, STBI_rgb_alpha);

    Texture sky(skyData, skyWidth, skyHeight);
    renderer.setSky(sky);

    stbi_image_free(skyData);

    // Draw call
    renderer.clear();

    mat4<float> model = rotationX<float>(M_PI / 5) * rotationY<float>(M_PI / 5);
    mat4<float> view = scale<float>(vec3<float>(0.9f));

    Uniforms uniforms(model, view);
    renderer.setUniforms(uniforms);

    renderer.draw(vertexBuffer, indexBuffer);

    // CPU image
    uint8_t* bufferCPU = new uint8_t[renderer.getFrameBuffer().size];

    hipMemcpy(bufferCPU, renderer.getFrameBuffer().buff, renderer.getFrameBuffer().size, hipMemcpyDeviceToHost);
    stbi_write_png("output.png", width, height, STBI_rgb, bufferCPU, width * STBI_rgb);
    
    delete[] bufferCPU;

    return 0;
}