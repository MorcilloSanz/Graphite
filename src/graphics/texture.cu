#include "hip/hip_runtime.h"
#include "texture.cuh"

namespace gph
{

Texture::Texture(uint8_t* hData, size_t _width, size_t _height) 
    : width(_width), height(_height) {

    size_t imageSize = width * height * 4 * sizeof(uint8_t);

    // 1. Crear un Array 2D CUDA
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();
    hipArray_t cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height);

    // 2. Copiar los datos desde el host al Array CUDA
    hipMemcpy2DToArray(cuArray, 0, 0, hData, width * 4 * sizeof(uint8_t),
                        width * 4 * sizeof(uint8_t), height, hipMemcpyHostToDevice);

    // 3. Configurar Resource Descriptor
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    // 4. Configurar Texture Descriptor
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;  // Coordenadas fuera del rango se envuelven
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModeLinear;    // Interpolación lineal
    texDesc.readMode = hipReadModeNormalizedFloat; // Leer como flotantes normalizados
    texDesc.normalizedCoords = 1; // Coordenadas normalizadas

    // 4. Crear el objeto de textura
    hipError_t err = hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);
    if (err != hipSuccess) {
        printf("Couldn't create cudaTextureObject: %s\n", hipGetErrorString(err));
    }  
}

}