#include "hip/hip_runtime.h"
#include "texture.cuh"

namespace gph
{

Texture::Texture(uint8_t* hData, size_t width, size_t height) 
    : TextureBase(width, height) {

    size_t imageSize = width * height * 4 * sizeof(uint8_t);

    // 2D CUDA Array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();
    hipArray_t cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height);
    hipMemcpy2DToArray(cuArray, 0, 0, hData, width * 4 * sizeof(uint8_t),
                        width * 4 * sizeof(uint8_t), height, hipMemcpyHostToDevice);

    // Resource Descriptor
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    // Configure Texture Descriptor
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeNormalizedFloat;
    texDesc.normalizedCoords = 1;

    // Create texture object
    hipError_t err = hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);
    if (err != hipSuccess) {
        printf("Couldn't create cudaTextureObject: %s\n", hipGetErrorString(err));
    }  
}

Texture::~Texture() {

    if (texObj)
        hipDestroyTextureObject(texObj);

    if (resDesc.resType == hipResourceTypeArray && resDesc.res.array.array)
        hipFreeArray(resDesc.res.array.array);
}

TextureHDR::TextureHDR(float* hData, size_t width, size_t height) 
    : TextureBase(width, height) {

    size_t imageSize = width * height * 4 * sizeof(float);

    // 2D CUDA Array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
    hipArray_t cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height);
    hipMemcpy2DToArray(cuArray, 0, 0, hData, width * 4 * sizeof(float),
                        width * 4 * sizeof(float), height, hipMemcpyHostToDevice);

    // Resource Descriptor
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    // Configure Texture Descriptor
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModeLinear;

    // **Cambio importante:** Leer como float directamente
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    // Create texture object
    hipError_t err = hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);
    if (err != hipSuccess) {
        printf("Couldn't create cudaTextureObject: %s\n", hipGetErrorString(err));
    }
}

TextureHDR::~TextureHDR() {

    if (texObj)
        hipDestroyTextureObject(texObj);

    if (resDesc.resType == hipResourceTypeArray && resDesc.res.array.array)
        hipFreeArray(resDesc.res.array.array);
}

}