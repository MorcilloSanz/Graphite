#include "buffer.cuh"

namespace gph
{

void check_cuda_error() {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << hipGetErrorString(error) << std::endl;
        exit(-1);
    }
}

template <typename T>
Buffer<T>::Buffer(size_t _size)
    : size(_size) {
    hipMalloc((void**)&buff, size);
    hipMemset(buff, 0, size);
}

template <typename T>
Buffer<T>::Buffer(T* data, size_t size) 
    : Buffer(size) {
    hipMemcpy(buff, data, size, hipMemcpyHostToDevice);
}

template <typename T>
Buffer<T>::~Buffer() {
    if(buff) hipFree(buff);
}

FrameBuffer::FrameBuffer(unsigned int _width, unsigned int _height)
    : Buffer<uint8_t>(_width * _height * 3), width(_width), height(_height) {
}

void FrameBuffer::clear() {
    if(buff) {
        hipMemset(buff, 0, size);
    }
}

template class Buffer<uint8_t>;
template class Buffer<char>;
template class Buffer<float>;
template class Buffer<double>;
template class Buffer<unsigned int>;
template class Buffer<int>;
template class Buffer<long>;

}