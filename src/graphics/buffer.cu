#include "buffer.cuh"

#include "kernel/fragment.cuh"
#include "kernel/vertex.cuh"

namespace gph
{

//------------------//
//  BufferRegister  //
//------------------//

BufferRegister* BufferRegister::getInstance() {
    if(instance == nullptr)
        instance = new BufferRegister();

    return instance;
}

void BufferRegister::destroyInstance() {
    if (instance != nullptr) {
        delete instance;
        instance = nullptr;
    }
}

BufferRegister* BufferRegister::instance = nullptr;

//-----------------//
//     Renderer    //
//-----------------//

void Renderer::init() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
}

void Renderer::destroy() {
    BufferRegister::destroyInstance();
}

void Renderer::draw() {

    BufferRegister* bufferRegister = BufferRegister::getInstance();
    if(bufferRegister->getBindedFrameBufferID() > 0) {

        // Kernel Frame Buffer
        Ptr<FrameBuffer> bindedFrameBuffer = bufferRegister->getBindedFrameBuffer();
        const unsigned int width = bindedFrameBuffer->getWidth();
        const unsigned int height = bindedFrameBuffer->getHeight();

        KernelFrameBuffer kernelFrameBuffer((uint8_t*)bindedFrameBuffer->getBuffer(), width, height);
        
        // Kernel Vertex Buffer
        size_t vertexBufferSize = 0;
        void* vertexBuffer = nullptr;

        if(bufferRegister->getBindedVertexBufferID() > 0) {
            Ptr<VertexBuffer> bindedVertexBuffer = bufferRegister->getBindedVertexBuffer();
            vertexBuffer = bindedVertexBuffer->getBuffer();
            vertexBufferSize = bindedVertexBuffer->getSize();
        }

        KernelBuffer kernelVertexBuffer(vertexBuffer, vertexBufferSize / sizeof(float));

        // Index Buffer
        size_t indexBufferSize = 0;
        void* indexBuffer = nullptr;

        if(bufferRegister->getBindedIndexBufferID() > 0) {
            Ptr<IndexBuffer> bindedIndexBuffer = bufferRegister->getBindedIndexBuffer();
            indexBuffer = bindedIndexBuffer->getBuffer();
            indexBufferSize = bindedIndexBuffer->getSize();
        }

        KernelBuffer kernelIndexBuffer(indexBuffer, indexBufferSize / sizeof(unsigned int));

        // Vertex kernel -> transform each vertex
        mat4<float> modelViewMatrix = uniforms.viewMatrix * uniforms.modelMatrix;

        int threadsPerBlockVertex = 256;
        int numBlocksVertex = (kernelIndexBuffer.count + threadsPerBlockVertex - 1) / threadsPerBlockVertex;

        kernel_vertex<<<numBlocksVertex, threadsPerBlockVertex>>>(kernelVertexBuffer, kernelIndexBuffer, modelViewMatrix);
        hipDeviceSynchronize();
  
        // Fragment kernel -> compute each fragment
        dim3 threadsPerBlock(16, 16);
        dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                        (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

        kernel_fragment<<<blocksPerGrid, threadsPerBlock>>>(kernelFrameBuffer, kernelVertexBuffer, kernelIndexBuffer);
        hipDeviceSynchronize();
    }
}

void Renderer::clear() {

    BufferRegister* bufferRegister = BufferRegister::getInstance();

    if(bufferRegister->getBindedFrameBufferID() > 0) {

        Ptr<FrameBuffer> bindedFrameBuffer = bufferRegister->getBindedFrameBuffer();
        bindedFrameBuffer->clear();
    }
}


//-------------//
//   Buffer    //
//-------------//

Buffer::Buffer(unsigned int _id, size_t _size)
    : id(_id), size(_size) {
    hipMalloc((void**)&buffer, size);
    check_cuda_error("Buffer::Buffer hipMalloc");
    hipMemset(buffer, 0, size);
    check_cuda_error("Buffer::Buffer hipMemset");
}

Buffer::~Buffer() {
    hipFree(buffer);
    check_cuda_error("Buffer::~Buffer hipFree");
}

Buffer::Buffer(const Buffer& buff)
    : id(buff.id), size(buff.size) {
    if (buff.buffer) {
        hipMalloc((void**)&buffer, size);
        check_cuda_error("Buffer::Buffer (copy) hipMalloc");
        hipMemcpy(buffer, buff.buffer, size, hipMemcpyDeviceToDevice);
        check_cuda_error("Buffer::Buffer (copy) hipMemcpy");
    }
}

Buffer::Buffer(Buffer&& buff) noexcept
    : buffer(buff.buffer), id(buff.id), size(buff.size) {
    buff.buffer = nullptr;
    buff.size = 0;
    buff.id = 0;
}

Buffer& Buffer::operator=(const Buffer& buff) {

    if (this != &buff) {
 
        hipFree(buffer);
        check_cuda_error("Buffer::operator= hipFree");


        id = buff.id;
        size = buff.size;

        if (buff.buffer) {
            hipMalloc((void**)&buffer, size);
            check_cuda_error("Buffer::operator= hipMalloc");
            hipMemcpy(buffer, buff.buffer, size, hipMemcpyDeviceToDevice);
            check_cuda_error("Buffer::operator= hipMemcpy");
        } else {
            buffer = nullptr;
        }
    }

    return *this;
}

Buffer& Buffer::operator=(Buffer&& buff) noexcept {

    if (this != &buff) {

        hipFree(buffer);
        check_cuda_error("Buffer::operator= (move) hipFree");

        buffer = buff.buffer;
        id = buff.id;
        size = buff.size;

        buff.buffer = nullptr;
        buff.size = 0;
        buff.id = 0;
    }

    return *this;
}

//-----------------//
//   FrameBuffer   //
//-----------------//

FrameBuffer::FrameBuffer(unsigned int id, unsigned int _width, unsigned int _height)
    : Buffer(id, 1 * width * height * 3), width(_width), height(_height) {
}

FrameBuffer::FrameBuffer(unsigned int _width, unsigned int _height)
    : FrameBuffer(0, width, height) {
}

FrameBuffer::FrameBuffer(const FrameBuffer& frameBuffer)
    : Buffer(frameBuffer), width(frameBuffer.width),
    height(frameBuffer.height) {
}

FrameBuffer::FrameBuffer(FrameBuffer&& frameBuffer) noexcept
    : Buffer(std::move(frameBuffer)), width(frameBuffer.width),
    height(frameBuffer.height) {
    frameBuffer.width = 0;
    frameBuffer.height = 0;
}

FrameBuffer& FrameBuffer::operator=(const FrameBuffer& frameBuffer) {

    if (this != &frameBuffer) {

        Buffer::operator=(frameBuffer);

        width = frameBuffer.width;
        height = frameBuffer.height;
    }

    return *this;
}

FrameBuffer& FrameBuffer::operator=(FrameBuffer&& frameBuffer) noexcept {

    if (this != &frameBuffer) {

        Buffer::operator=(std::move(frameBuffer));

        width = frameBuffer.width;
        height = frameBuffer.height;

        frameBuffer.width = 0;
        frameBuffer.height = 0;
    }

    return *this;
}

Ptr<FrameBuffer> FrameBuffer::New(unsigned int width, unsigned int height) {

    BufferRegister* bufferRegister = BufferRegister::getInstance();

    int id = bufferRegister->getFrameBuffers().size() + 1;
    Ptr<FrameBuffer> frameBuffer = std::make_shared<FrameBuffer>(id, width, height);
    bufferRegister->addFrameBuffer(frameBuffer);

    return frameBuffer;
}

void FrameBuffer::bind() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
    bufferRegister->bindFbo(id);
}

void FrameBuffer::unbind() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
    bufferRegister->bindFbo(0);
}

void FrameBuffer::clear() {
    hipMemset(buffer, 0, size);
}

//------------------//
//   VertexBuffer   //
//------------------//

VertexBuffer::VertexBuffer(unsigned int id, float* data, size_t size) 
    : Buffer(id, size) {
    hipMemcpy(buffer, data, size, hipMemcpyHostToDevice);
    check_cuda_error("VertexBuffer::VertexBuffer hipMemcpy");
}

VertexBuffer::VertexBuffer(const VertexBuffer& vertexBuffer) 
    : Buffer(vertexBuffer) {
}

VertexBuffer::VertexBuffer(VertexBuffer&& vertexBuffer) noexcept 
    : Buffer(std::move(vertexBuffer)) {
}

VertexBuffer& VertexBuffer::operator=(const VertexBuffer& vertexBuffer) {

    if(this != &vertexBuffer) {
        Buffer::operator=(vertexBuffer);
    }

    return *this;
}

VertexBuffer& VertexBuffer::operator=(VertexBuffer&& vertexBuffer) noexcept {

    if(this != &vertexBuffer) {
        Buffer::operator=(std::move(vertexBuffer));
    }

    return *this;
}

Ptr<VertexBuffer> VertexBuffer::New(float* data, size_t size) {

    BufferRegister* bufferRegister = BufferRegister::getInstance();

    int id = bufferRegister->getVertexBuffers().size() + 1;
    Ptr<VertexBuffer> vertexBuffer = std::make_shared<VertexBuffer>(id, data, size);
    bufferRegister->addVertexBuffer(vertexBuffer);

    return vertexBuffer;
}

void VertexBuffer::bind() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
    bufferRegister->bindVbo(id);
}

void VertexBuffer::unbind() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
    bufferRegister->bindVbo(0);
}

//-----------------//
//   IndexBuffer   //
//-----------------//

IndexBuffer::IndexBuffer(unsigned int id, unsigned int* indices, size_t size) 
    : Buffer(id, size) {
    hipMemcpy(buffer, indices, size, hipMemcpyHostToDevice);
    check_cuda_error("IndexBuffer::IndexBuffer hipMemcpy");
}

IndexBuffer::IndexBuffer(const IndexBuffer& indexBuffer) 
    : Buffer(indexBuffer) {
}

IndexBuffer::IndexBuffer(IndexBuffer&& indexBuffer) noexcept 
    : Buffer(std::move(indexBuffer)) {
}

IndexBuffer& IndexBuffer::operator=(const IndexBuffer& indexBuffer) {

    if(this != &indexBuffer) {
        Buffer::operator=(indexBuffer);
    }

    return *this;
}

IndexBuffer& IndexBuffer::operator=(IndexBuffer&& indexBuffer) noexcept {

    if(this != &indexBuffer) {
        Buffer::operator=(std::move(indexBuffer));
    }

    return *this;
}

Ptr<IndexBuffer> IndexBuffer::New(unsigned int* indices, size_t size) {

    BufferRegister* bufferRegister = BufferRegister::getInstance();

    int id = bufferRegister->getIndexBuffers().size() + 1;
    Ptr<IndexBuffer> indexBuffer = std::make_shared<IndexBuffer>(id, indices, size);
    bufferRegister->addIndexBuffer(indexBuffer);

    return indexBuffer;
}

void IndexBuffer::bind() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
    bufferRegister->bindIbo(id);
}

void IndexBuffer::unbind() {
    BufferRegister* bufferRegister = BufferRegister::getInstance();
    bufferRegister->bindIbo(0);
}

//------------//
//    CUDA    //
//------------//

void check_cuda_error(const char* message) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error after " << message << ": " << hipGetErrorString(error) << std::endl;
        exit(-1);
    }
}

}