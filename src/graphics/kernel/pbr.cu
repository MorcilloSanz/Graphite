#include "hip/hip_runtime.h"
#include "pbr.cuh"

namespace gph
{

__device__ float clamp(float value, float minVal, float maxVal) {
    return max(minVal, min(value, maxVal));
}

__device__ void orthonormalBasis(vec3<float> N, vec3<float>& tangent, vec3<float>& bitangent) {
    if (fabs(N.x) > fabs(N.z))
        tangent = vec3<float>(-N.y, N.x, 0.0f).normalize();
    else
        tangent = vec3<float>(0.0f, -N.z, N.y).normalize();

    bitangent = N.cross(tangent);
}

__device__ vec3<float> sampleGGX(vec3<float> N, vec3<float> V, float roughness, hiprandState& state) {
    float alpha = roughness * roughness;

    // Generamos dos números aleatorios en [0,1]
    float xi1 = hiprand_uniform(&state);
    float xi2 = hiprand_uniform(&state);

    // Muestreo de GGX para theta_h y phi_h
    float theta_h = atan(sqrt(alpha * alpha * xi1) / sqrt(1.0 - xi1));
    float phi_h = 2.0 * M_PI * xi2;

    // Convertir a coordenadas cartesianas
    float sinTheta = sin(theta_h);
    float cosTheta = cos(theta_h);
    float sinPhi = sin(phi_h);
    float cosPhi = cos(phi_h);

    // Vector mitad H en espacio local
    vec3<float> H = vec3<float>(sinTheta * cosPhi, sinTheta * sinPhi, cosTheta);

    // Convertir H al sistema de referencia de la normal N
    vec3<float> tangent, bitangent;
    orthonormalBasis(N, tangent, bitangent);
    H = (tangent * H.x + bitangent * H.y + N * H.z).normalize();

    // Reflexión especular para obtener L
    vec3<float> L = (H * V.dot(H) * 2.0f - V).normalize();

    return L;
}

__device__ float distributionGGX(vec3<float> N, vec3<float> H, float roughness) {

    float a = roughness * roughness;
    float a2 = a * a;
    float NdotH = max(N.dot(H), 0.0);
    float NdotH2 = NdotH * NdotH;

    float nom   = a2;
    float denom = (NdotH2 * (a2 - 1.0) + 1.0);
    denom = M_PI * denom * denom;

    return nom / denom;
}

__device__ float geometrySchlickGGX(float NdotV, float roughness) {

    float r = (roughness + 1.0);
    float k = (r * r) / 8.0;

    float nom   = NdotV;
    float denom = NdotV * (1.0 - k) + k;

    return nom / denom;
}

__device__ float geometrySmith(vec3<float> N, vec3<float> V, vec3<float> L, float roughness) {

    float NdotV = max(N.dot(V), 0.0);
    float NdotL = max(N.dot(L), 0.0);
    float ggx2 = geometrySchlickGGX(NdotV, roughness);
    float ggx1 = geometrySchlickGGX(NdotL, roughness);

    return ggx1 * ggx2;
}

__device__ vec3<float> fresnelSchlick(float cosTheta, vec3<float> F0) {
    return F0 + (vec3<float>(1.0f) - F0) * pow(clamp(1.0 - cosTheta, 0.0, 1.0), 5.0);
}

__device__ vec3<float> specularCookTorrance(vec3<float> H, vec3<float> normal, vec3<float> wo, vec3<float> wi, vec3<float> F, float roughness) {
    
    float NDF = distributionGGX(normal, H, roughness);   
    float G = geometrySmith(normal, wo, wi, roughness);      
    
    vec3<float> numerator = F * NDF * G; 
    float denominator = 4.0 * normal.dot(wo) * normal.dot(wi) + 0.0001;
    vec3<float> specular = numerator / denominator;
    
    return specular;
}
    
}