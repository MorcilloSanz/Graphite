#include "renderer.cuh"

#include "buffer.cuh"
#include "texture.cuh"

#include "kernel/fragment.cuh"
#include "kernel/vertex.cuh"

namespace gph
{

Renderer::Renderer(unsigned int width, unsigned int height) 
    : frameBuffer(FrameBuffer::New(width, height)), hasSky(false) {
}

KernelVertexParams Renderer::getKernelVertexParams(Scene::Ptr scene) {

    KernelVertexParams params;

    Buffer<float>::Ptr vertexBuffer = scene->vertexBuffer;
    Buffer<unsigned int>::Ptr indexBuffer = scene->indexBuffer;

    KernelVertexBuffer kernelVertexBuffer(vertexBuffer->buff, vertexBuffer->size);
    params.vertexBuffer = kernelVertexBuffer;

    KernelIndexBuffer kernelIndexBuffer(indexBuffer->buff, indexBuffer->size);
    params.indexBuffer = kernelIndexBuffer;

    mat4<float> modelviewMatrix = uniforms.viewMatrix * uniforms.modelMatrix;
    params.modelviewMatrix = modelviewMatrix;

    mat3<float> normalMatrix;
    normalMatrix.row1 = uniforms.modelMatrix.row1.xyz();
    normalMatrix.row2 = uniforms.modelMatrix.row2.xyz();
    normalMatrix.row3 = uniforms.modelMatrix.row3.xyz();
    normalMatrix = normalMatrix.inverse().transpose();
    params.normalMatrix = normalMatrix;

    return params;
}

void Renderer::vertexShader(Scene::Ptr scene) {

    KernelVertexParams params = getKernelVertexParams(scene);

    int threadsPerBlock = 256;
    int count = scene->indexBuffer->size / sizeof(unsigned int);
    int numBlocks = (count + threadsPerBlock - 1) / threadsPerBlock;

    kernel_vertex<<<numBlocks, threadsPerBlock>>>(params);
    hipDeviceSynchronize();
}

KernelFragmentParams Renderer::getKernelFragmentParams(Scene::Ptr scene) {

    KernelFragmentParams params;

    Buffer<float>::Ptr vertexBuffer = scene->vertexBuffer;
    Buffer<unsigned int>::Ptr indexBuffer = scene->indexBuffer;

    KernelFrameBuffer kernelFrameBuffer(frameBuffer->buff, frameBuffer->width, frameBuffer->height);
    params.frameBuffer = kernelFrameBuffer;

    KernelVertexBuffer kernelVertexBuffer(vertexBuffer->buff, vertexBuffer->size);
    params.vertexBuffer = kernelVertexBuffer;

    KernelIndexBuffer kernelIndexBuffer(indexBuffer->buff, indexBuffer->size);
    params.indexBuffer = kernelIndexBuffer;

    KernelTexture kernelSky(sky->getTextureObject(), hasSky);
    params.sky = kernelSky;

    params.materialsCount = 0;

    return params;
}

void Renderer::fragmentShader(Scene::Ptr scene) {

    KernelFragmentParams params = getKernelFragmentParams(scene);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((frameBuffer->width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (frameBuffer->height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    kernel_fragment<<<blocksPerGrid, threadsPerBlock>>>(params);
    hipDeviceSynchronize();
}

void Renderer::setSky(Texture::Ptr sky) {
    this->sky = sky;
    hasSky = true;
}

void Renderer::draw(Scene::Ptr scene) {
    vertexShader(scene);
    fragmentShader(scene);
}

void Renderer::clear() {
    frameBuffer->clear();
}

}