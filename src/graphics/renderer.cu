#include "renderer.cuh"

#include "buffer.cuh"
#include "texture.cuh"

#include "kernel/fragment.cuh"
#include "kernel/vertex.cuh"

namespace gph
{

void Renderer::vertexShader(const Buffer<float>& vertexBuffer, const Buffer<unsigned int>& indexBuffer) {

    mat4<float> modelViewMatrix = uniforms.viewMatrix * uniforms.modelMatrix;

    int threadsPerBlockVertex = 256;
    int count = indexBuffer.size / sizeof(unsigned int);
    int numBlocksVertex = (count + threadsPerBlockVertex - 1) / threadsPerBlockVertex;

    kernel_vertex<<<numBlocksVertex, threadsPerBlockVertex>>>(vertexBuffer.buff, vertexBuffer.size, 
        indexBuffer.buff, indexBuffer.size, modelViewMatrix);

    hipDeviceSynchronize();
}

void Renderer::fragmentShader(const Buffer<float>& vertexBuffer, const Buffer<unsigned int>& indexBuffer) {

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((frameBuffer.width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (frameBuffer.height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    kernel_fragment<<<blocksPerGrid, threadsPerBlock>>>(frameBuffer.buff, frameBuffer.width, frameBuffer.height, vertexBuffer.buff, 
        vertexBuffer.size, indexBuffer.buff, indexBuffer.size);
        
    hipDeviceSynchronize();
}

void Renderer::draw(const Buffer<float>& vertexBuffer, const Buffer<unsigned int>& indexBuffer) {
    vertexShader(vertexBuffer, indexBuffer);
    fragmentShader(vertexBuffer, indexBuffer);
}

void Renderer::clear() {
    frameBuffer.clear();
}

}